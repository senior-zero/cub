#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockAdjacentDifference utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <limits>
#include <typeinfo>
#include <memory>

#include <cub/util_allocator.cuh>
#include <cub/block/block_adjacent_difference.cuh>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/mismatch.h>
#include <thrust/tabulate.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/sort.h>

#include "test_util.h"

using namespace hipcub;


/**
 * \brief Generates integer sequence \f$S_n=i(i-1)/2\f$.
 *
 * The adjacent difference of this sequence produce consecutive numbers:
 * \f[
 *   p = \frac{i(i - 1)}{2} \\
 *   n = \frac{(i + 1) i}{2} \\
 *   n - p = i \\
 *   \frac{(i + 1) i}{2} - \frac{i (i - 1)}{2} = i \\
 *   (i + 1) i - i (i - 1) = 2 i \\
 *   (i + 1) - (i - 1) = 2 \\
 *   2 = 2
 * \f]
 */
template <typename DestT>
struct TestSequenceGenerator
{
  template <typename SourceT>
  __device__ __host__ DestT operator()(SourceT index) const
  {
    return static_cast<DestT>(index * (index - 1) / SourceT(2));
  }
};



struct CustomType
{
  unsigned int key;
  unsigned int value;

  __device__ __host__ CustomType()
    : key(0)
    , value(0)
  {}

  __device__ __host__ CustomType(unsigned int key, unsigned int value)
    : key(key)
    , value(value)
  {}
};


__device__ __host__ bool operator==(const CustomType& lhs,
                                    const CustomType& rhs)
{
  return lhs.key == rhs.key && lhs.value == rhs.value;
}

__device__ __host__ bool operator!=(const CustomType& lhs,
                                    const CustomType& rhs)
{
  return !(lhs == rhs);
}

__device__ __host__ CustomType operator-(const CustomType& lhs,
                                         const CustomType& rhs)
{
  return CustomType{lhs.key - rhs.key, lhs.value - rhs.value};
}

struct CustomDifference
{
  template <typename DataType>
  __device__ DataType operator()(DataType &lhs, DataType &rhs)
  {
    return lhs - rhs;
  }
};


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void
BlockAdjacentDifferenceLastTileTestKernel(const DataType *input,
                                          DataType *output,
                                          unsigned int valid_items)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage).SubtractLeftPartialTile(
      thread_result,
      thread_data,
      CustomDifference(),
      valid_items);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage).SubtractRightPartialTile(
      thread_result,
      thread_data,
      CustomDifference(),
      valid_items);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_result[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void BlockAdjacentDifferenceTestKernel(DataType *data)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = data[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_result,
                    thread_data,
                    CustomDifference());
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_result,
                     thread_data,
                     CustomDifference());
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    data[thread_offset + item] = thread_result[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void
BlockAdjacentDifferenceLastTileTestInplaceKernel(const DataType *input,
                                                 DataType *output,
                                                 unsigned int valid_items)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeftPartialTile(thread_data,
                               thread_data,
                               CustomDifference(),
                               valid_items);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRightPartialTile(thread_data,
                                thread_data,
                                CustomDifference(),
                                valid_items);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_data[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void
BlockAdjacentDifferenceTestInplaceKernel(DataType *data)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = data[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data,
                    thread_data,
                    CustomDifference());
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data,
                     thread_data,
                     CustomDifference());
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    data[thread_offset + item] = thread_data[item];
  }
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void BlockAdjacentDifferenceLastTileTest(const DataType *input,
                                         DataType *output,
                                         unsigned int valid_items)
{
  BlockAdjacentDifferenceLastTileTestKernel<DataType,
                                            ThreadsInBlock,
                                            ItemsPerThread,
                                            ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, valid_items);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void BlockAdjacentDifferenceTest(DataType *data)
{
  BlockAdjacentDifferenceTestKernel<DataType,
                                    ThreadsInBlock,
                                    ItemsPerThread,
                                    ReadLeft><<<1, ThreadsInBlock>>>(data);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void BlockAdjacentDifferenceLastTileInplaceTest(const DataType *input,
                                                DataType *output,
                                                unsigned int valid_items)
{
  BlockAdjacentDifferenceLastTileTestInplaceKernel<DataType,
                                                   ThreadsInBlock,
                                                   ItemsPerThread,
                                                   ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, valid_items);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void BlockAdjacentDifferenceInplaceTest(DataType *data)
{
  BlockAdjacentDifferenceTestInplaceKernel<DataType,
                                           ThreadsInBlock,
                                           ItemsPerThread,
                                           ReadLeft>
    <<<1, ThreadsInBlock>>>(data);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename FirstIteratorT,
          typename SecondOperatorT>
bool CheckResult(FirstIteratorT first_begin,
                 FirstIteratorT first_end,
                 SecondOperatorT second_begin)
{
  auto err = thrust::mismatch(first_begin, first_end, second_begin);

  if (err.first != first_end)
  {
    return false;
  }

  return true;
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestLastTile(bool inplace,
                  unsigned int num_items,
                  thrust::device_vector<DataType> &d_input)
{
  thrust::tabulate(d_input.begin(),
                   d_input.end(),
                   TestSequenceGenerator<DataType>{});
  thrust::device_vector<DataType> d_output(d_input.size());

  constexpr bool read_left = true;
  constexpr bool read_right = false;

  DataType *d_input_ptr = thrust::raw_pointer_cast(d_input.data());
  DataType *d_output_ptr = thrust::raw_pointer_cast(d_output.data());

  if (inplace)
  {
    BlockAdjacentDifferenceLastTileInplaceTest<DataType,
                                               ItemsPerThread,
                                               ThreadsInBlock,
                                               read_left>(d_input_ptr,
                                                          d_output_ptr,
                                                          num_items);
  }
  else
  {
    BlockAdjacentDifferenceLastTileTest<DataType,
                                        ItemsPerThread,
                                        ThreadsInBlock,
                                        read_left>(d_input_ptr,
                                                   d_output_ptr,
                                                   num_items);
  }

  {
    using CountingIteratorT =
      typename thrust::counting_iterator<DataType,
        thrust::use_default,
        std::size_t,
        std::size_t>;

    AssertEquals(d_output.front(), d_input.front());
    AssertTrue(CheckResult(d_output.begin() + 1,
                           d_output.begin() + num_items,
                           CountingIteratorT(DataType{0})));
    AssertTrue(CheckResult(d_output.begin() + num_items,
                           d_output.end(),
                           d_input.begin() + num_items));
  }


  thrust::tabulate(d_input.begin(),
                   d_input.end(),
                   TestSequenceGenerator<DataType>{});

  if (inplace)
  {
    BlockAdjacentDifferenceLastTileInplaceTest<DataType,
                                               ItemsPerThread,
                                               ThreadsInBlock,
                                               read_right>(d_input_ptr,
                                                           d_output_ptr,
                                                           num_items);
  }
  else
  {
    BlockAdjacentDifferenceLastTileTest<DataType,
                                        ItemsPerThread,
                                        ThreadsInBlock,
                                        read_right>(d_input_ptr,
                                                    d_output_ptr,
                                                    num_items);
  }

  {
    thrust::device_vector<DataType> reference(num_items);
    thrust::sequence(reference.begin(),
                     reference.end(),
                     static_cast<DataType>(0),
                     static_cast<DataType>(-1));

    AssertTrue(CheckResult(d_output.begin(),
                           d_output.begin() + num_items - 1,
                           reference.begin()));
    AssertTrue(CheckResult(d_output.begin() + num_items - 1,
                           d_output.end(),
                           d_input.begin() + num_items - 1));
  }
}

struct IntToCustomType
{
  unsigned int offset;

  IntToCustomType()
      : offset(0)
  {}

  explicit IntToCustomType(unsigned int offset)
      : offset(offset)
  {}

  __device__ __host__ CustomType operator()(unsigned int idx) const
  {
    return { idx + offset, idx + offset };
  }
};


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestFullTile(bool inplace,
                  thrust::device_vector<DataType> &d_data)
{
  thrust::tabulate(d_data.begin(),
                   d_data.end(),
                   TestSequenceGenerator<DataType>{});

  constexpr bool read_left  = true;
  constexpr bool read_right = false;

  DataType *d_data_ptr = thrust::raw_pointer_cast(d_data.data());

  if (inplace)
  {
    BlockAdjacentDifferenceInplaceTest<DataType,
                                       ItemsPerThread,
                                       ThreadsInBlock,
                                       read_left>(d_data_ptr);
  }
  else
  {
    BlockAdjacentDifferenceTest<DataType,
                                ItemsPerThread,
                                ThreadsInBlock,
                                read_left>(d_data_ptr);
  }

  {
    using CountingIteratorT =
    typename thrust::counting_iterator<DataType,
      thrust::use_default,
      std::size_t,
      std::size_t>;

    AssertEquals(d_data.front(), TestSequenceGenerator<DataType>{}(0));
    AssertTrue(CheckResult(d_data.begin() + 1,
                           d_data.end(),
                           CountingIteratorT(DataType{0})));
  }

  thrust::tabulate(d_data.begin(),
                   d_data.end(),
                   TestSequenceGenerator<DataType>{});

  if (inplace)
  {
    BlockAdjacentDifferenceInplaceTest<DataType,
                                       ItemsPerThread,
                                       ThreadsInBlock,
                                       read_right>(d_data_ptr);
  }
  else
  {
    BlockAdjacentDifferenceTest<DataType,
                                ItemsPerThread,
                                ThreadsInBlock,
                                read_right>(d_data_ptr);
  }

  {
    thrust::device_vector<DataType> reference(d_data.size());
    thrust::sequence(reference.begin(),
                     reference.end(),
                     static_cast<DataType>(0),
                     static_cast<DataType>(-1));

    AssertTrue(CheckResult(d_data.begin(),
                           d_data.end() - 1,
                           reference.begin()));
    AssertEquals(d_data.back(),
                 TestSequenceGenerator<DataType>{}(d_data.size() - 1));
  }
}


template <unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestCustomType(bool inplace,
                    thrust::device_vector<CustomType> &d_data)
{
  thrust::tabulate(d_data.begin(), d_data.end(), IntToCustomType{1});
  CustomType *d_data_ptr = thrust::raw_pointer_cast(d_data.data());

  constexpr bool read_left  = true;
  constexpr bool read_right = false;

  if (inplace)
  {
    BlockAdjacentDifferenceInplaceTest<CustomType,
                                       ItemsPerThread,
                                       ThreadsInBlock,
                                       read_left>(d_data_ptr);
  }
  else
  {
    BlockAdjacentDifferenceTest<CustomType,
                                ItemsPerThread,
                                ThreadsInBlock,
                                read_left>(d_data_ptr);
  }

  {
    const std::size_t expected_count = d_data.size();
    const std::size_t actual_count =
      thrust::count(d_data.begin(), d_data.end(), CustomType{1, 1});

    AssertEquals(expected_count, actual_count);
  }

  thrust::tabulate(d_data.begin(), d_data.end(), IntToCustomType{});

  if (inplace)
  {
    BlockAdjacentDifferenceInplaceTest<CustomType,
                                       ItemsPerThread,
                                       ThreadsInBlock,
                                       read_right>(d_data_ptr);
  }
  else
  {
    BlockAdjacentDifferenceTest<CustomType,
      ItemsPerThread,
      ThreadsInBlock,
      read_right>(d_data_ptr);
  }

  {
    const auto unsigned_minus_one = static_cast<unsigned int>(-1);

    const std::size_t expected_count = d_data.size() - 1;
    const std::size_t actual_count =
      thrust::count(d_data.begin(),
                    d_data.end() - 1,
                    CustomType{unsigned_minus_one, unsigned_minus_one});

    AssertEquals(expected_count, actual_count);
  }
}


template <
  typename ValueType,
  unsigned int ItemsPerThread,
  unsigned int ThreadsInBlock>
void Test(bool inplace)
{
  constexpr int tile_size = ItemsPerThread * ThreadsInBlock;
  thrust::device_vector<ValueType> d_values(tile_size);

  for (unsigned int num_items = tile_size; num_items > 1; num_items /= 2)
  {
    TestLastTile<ValueType, ItemsPerThread, ThreadsInBlock>(inplace,
                                                            num_items,
                                                            d_values);
  }

  TestFullTile<ValueType, ItemsPerThread, ThreadsInBlock>(inplace, d_values);
}


template <unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestCustomType(bool inplace)
{
  constexpr int tile_size = ItemsPerThread * ThreadsInBlock;
  thrust::device_vector<CustomType> d_values(tile_size);
  TestCustomType<ItemsPerThread, ThreadsInBlock>(inplace, d_values);
}


template <unsigned int ItemsPerThread, unsigned int ThreadsPerBlock>
void Test(bool inplace)
{
  Test<std::uint8_t,  ItemsPerThread, ThreadsPerBlock>(inplace);
  Test<std::uint16_t, ItemsPerThread, ThreadsPerBlock>(inplace);
  Test<std::uint32_t, ItemsPerThread, ThreadsPerBlock>(inplace);
  Test<std::uint64_t, ItemsPerThread, ThreadsPerBlock>(inplace);
}


template <unsigned int ItemsPerThread>
void Test(bool inplace)
{
  Test<ItemsPerThread, 32>(inplace);
  Test<ItemsPerThread, 256>(inplace);
}


template <unsigned int ItemsPerThread>
void Test()
{
  Test<ItemsPerThread>(false);
  Test<ItemsPerThread>(true);
}


int main(int argc, char** argv)
{
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  Test<1>();
  Test<2>();
  Test<10>();
  Test<15>();

  // More of a compilation check
  TestCustomType<5, 256>(true);

  return 0;
}
