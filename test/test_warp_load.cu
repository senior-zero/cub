#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>
#include <stdio.h>

#include <cub/warp/warp_load.cuh>
#include <cub/iterator/cache_modified_input_iterator.cuh>
#include <cub/iterator/cache_modified_output_iterator.cuh>
#include <cub/iterator/discard_output_iterator.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

using namespace hipcub;

__global__ void kernel(const int *input, int *output)
{
  constexpr int block_size = 256;
  constexpr int warp_size = 4;
  constexpr int items_per_thread = 4;

  using WarpLoadT = WarpLoad<int,
                             items_per_thread,
                             WarpLoadAlgorithm::WARP_LOAD_TRANSPOSE,
                             warp_size>;

  __shared__
    typename WarpLoadT::TempStorage temp_storage[block_size / warp_size];

  int reg[items_per_thread];
  WarpLoadT(temp_storage[threadIdx.x / warp_size]).Load(input, reg);

  for (int i = 0; i < items_per_thread; i++)
  {
    output[i] = reg[i];
  }
}


int main(int argc, char** argv)
{
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());
}



