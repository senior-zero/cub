#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>

#include <cub/warp/warp_load.cuh>
#include <cub/iterator/cache_modified_input_iterator.cuh>
#include <cub/iterator/cache_modified_output_iterator.cuh>
#include <cub/iterator/discard_output_iterator.cuh>
#include <cub/util_allocator.cuh>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include "test_util.h"

using namespace hipcub;

template <int                 BlockThreads,
          int                 WarpThreads,
          int                 ItemsPerThread,
          WarpLoadAlgorithm   LoadAlgorithm,
          typename            InputIteratorT>
__global__ void kernel(InputIteratorT input,
                       int *err)
{
  using InputT = typename std::iterator_traits<InputIteratorT>::value_type;

  using WarpLoadT = WarpLoad<InputT,
                             ItemsPerThread,
                             LoadAlgorithm,
                             WarpThreads>;

  constexpr int warps_in_block = BlockThreads / WarpThreads;
  const int warp_id = static_cast<int>(threadIdx.x) / WarpThreads;

  __shared__
    typename WarpLoadT::TempStorage temp_storage[warps_in_block];

  InputT reg[ItemsPerThread];
  WarpLoadT(temp_storage[warp_id]).Load(input, reg);

  for (int item = 0; item < ItemsPerThread; item++)
  {
    const auto expected_value =
      static_cast<InputT>(threadIdx.x * ItemsPerThread + item);

    if (reg[item] != expected_value)
    {
      atomicAdd(err, 1);
    }
  }
}

template <typename            T,
          int                 BlockThreads,
          int                 WarpThreads,
          int                 ItemsPerThread,
          WarpLoadAlgorithm   LoadAlgorithm>
void TestPointer()
{
  const int tile_size = WarpThreads * ItemsPerThread;
  const int total_warps = BlockThreads / WarpThreads;
  const int elements = total_warps * tile_size;

  thrust::device_vector<int> err(1);
  thrust::device_vector<T> input(elements);
  thrust::sequence(input.begin(), input.end());

  kernel<BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm>
    <<<1, BlockThreads>>>(thrust::raw_pointer_cast(input.data()),
                          thrust::raw_pointer_cast(err.data()));
  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());

  AssertEquals(input[0], 0);
}

template <typename            T,
          int                 BlockThreads,
          int                 WarpThreads,
          int                 ItemsPerThread,
          WarpLoadAlgorithm   LoadAlgorithm,
          CacheLoadModifier   LoadModifier>
void TestIterator()
{
  const int tile_size = WarpThreads * ItemsPerThread;
  const int total_warps = BlockThreads / WarpThreads;
  const int elements = total_warps * tile_size;

  thrust::device_vector<int> err(1);
  thrust::device_vector<T> input(elements);
  thrust::sequence(input.begin(), input.end());

  kernel<BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm>
    <<<1, BlockThreads>>>(CacheModifiedInputIterator<LoadModifier, T>(
                            thrust::raw_pointer_cast(input.data())),
                          thrust::raw_pointer_cast(err.data()));
  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());

  AssertEquals(input[0], 0);
}

template <typename            T,
          int                 BlockThreads,
          int                 WarpThreads,
          int                 ItemsPerThread,
          WarpLoadAlgorithm   LoadAlgorithm>
void TestIterator()
{
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_DEFAULT>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_CA>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_CG>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_CS>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_CV>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_LDG>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm, CacheLoadModifier::LOAD_VOLATILE>();
}


template <typename            T,
          int                 BlockThreads,
          int                 WarpThreads,
          int                 ItemsPerThread,
          WarpLoadAlgorithm   LoadAlgorithm>
void Test()
{
  TestPointer<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm>();
  TestIterator<T, BlockThreads, WarpThreads, ItemsPerThread, LoadAlgorithm>();
}

template <typename  T,
          int       BlockThreads,
          int       WarpThreads,
          int       ItemsPerThread>
void Test()
{
  Test<T, BlockThreads, WarpThreads, ItemsPerThread, WarpLoadAlgorithm::WARP_LOAD_DIRECT>();
  Test<T, BlockThreads, WarpThreads, ItemsPerThread, WarpLoadAlgorithm::WARP_LOAD_STRIPED>();
  Test<T, BlockThreads, WarpThreads, ItemsPerThread, WarpLoadAlgorithm::WARP_LOAD_TRANSPOSE>();
  Test<T, BlockThreads, WarpThreads, ItemsPerThread, WarpLoadAlgorithm::WARP_LOAD_VECTORIZE>();
}


template <typename T,
          int      BlockThreads,
          int      WarpThreads>
void Test()
{
  Test<T, BlockThreads, WarpThreads, 1>();
  Test<T, BlockThreads, WarpThreads, 4>();
  Test<T, BlockThreads, WarpThreads, 7>();
}

template <typename T,
          int BlockThreads>
void Test()
{
  Test<T, BlockThreads, 4>();
  Test<T, BlockThreads, 16>();
  Test<T, BlockThreads, 32>();
}

template <int BlockThreads>
void Test()
{
  Test<std::uint16_t, BlockThreads>();
  Test<std::uint32_t, BlockThreads>();
  Test<std::uint64_t, BlockThreads>();
}

int main(int argc, char** argv)
{
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  Test<256>();
}



