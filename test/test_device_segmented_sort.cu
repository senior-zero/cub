#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <fstream>

#include <cub/device/device_segmented_sort.cuh>
#include <test_util.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

using namespace hipcub;


constexpr static int MAX_ITERATIONS = 10;


class SizeGroupDescription
{
public:
  SizeGroupDescription(const int segments,
                       const int segment_size)
      : segments(segments)
      , segment_size(segment_size)
  {}

  int segments {};
  int segment_size {};
};

template <typename KeyT,
          typename OffsetT>
struct SegmentChecker
{
  const KeyT *sorted_keys {};
  const OffsetT *offsets {};

  explicit SegmentChecker(const KeyT *sorted_keys,
                          const OffsetT *offsets)
    : sorted_keys(sorted_keys)
    , offsets(offsets)
  {}

  __device__ bool operator()(unsigned int segment_id)
  {
    const unsigned int segment_begin = offsets[segment_id];
    const unsigned int segment_end = offsets[segment_id + 1];

    unsigned int counter = 0;
    for (unsigned int i = segment_begin; i < segment_end; i++)
    {
      if (sorted_keys[i] != static_cast<KeyT>(counter++))
      {
        return false;
      }
    }

    return true;
  }
};

template <typename KeyT,
          typename OffsetT>
struct DescendingSegmentChecker
{
  const KeyT *sorted_keys{};
  const OffsetT *offsets{};

  explicit DescendingSegmentChecker(const KeyT *sorted_keys,
                                    const OffsetT *offsets)
      : sorted_keys(sorted_keys)
      , offsets(offsets)
  {}

  __device__ bool operator()(unsigned int segment_id)
  {
    const int segment_begin = static_cast<int>(offsets[segment_id]);
    const int segment_end   = static_cast<int>(offsets[segment_id + 1]);

    unsigned int counter = 0;
    for (int i = segment_end - 1; i >= segment_begin; i--)
    {
      if (sorted_keys[i] != static_cast<KeyT>(counter++))
      {
        return false;
      }
    }

    return true;
  }
};

template <typename KeyT,
          typename OffsetT>
struct ReversedIOTA
{
  KeyT *data {};
  const OffsetT *offsets {};

  ReversedIOTA(KeyT *data,
               const OffsetT *offsets)
    : data(data)
    , offsets(offsets)
  {}

  __device__ void operator()(unsigned int segment_id) const
  {
    const unsigned int segment_begin = offsets[segment_id];
    const unsigned int segment_end = offsets[segment_id + 1];
    const unsigned int segment_size = segment_end - segment_begin;

    unsigned int count = 0;
    for (unsigned int i = segment_begin; i < segment_end; i++)
    {
      data[i] = static_cast<KeyT>(segment_size - 1 - count++);
    }
  }
};


template <typename KeyT,
          typename OffsetT>
struct IOTA
{
  KeyT *data{};
  const OffsetT *offsets{};

  IOTA(KeyT *data, const OffsetT *offsets)
      : data(data)
      , offsets(offsets)
  {}

  __device__ void operator()(unsigned int segment_id) const
  {
    const unsigned int segment_begin = offsets[segment_id];
    const unsigned int segment_end   = offsets[segment_id + 1];

    unsigned int count = 0;
    for (unsigned int i = segment_begin; i < segment_end; i++)
    {
      data[i] = static_cast<KeyT>(count++);
    }
  }
};


template <typename KeyT,
          typename OffsetT,
          typename ValueT = hipcub::NullType>
class Input
{
  thrust::default_random_engine random_engine;
  thrust::device_vector<OffsetT> d_segment_sizes;
  thrust::device_vector<OffsetT> d_offsets;
  thrust::host_vector<OffsetT> h_offsets;

  using MaskedValueT = typename std::conditional<
    std::is_same<ValueT, hipcub::NullType>::value,
    KeyT,
    ValueT>::type;

  bool reverse {};
  OffsetT num_items {};
  thrust::device_vector<KeyT> d_keys;
  thrust::device_vector<MaskedValueT> d_values;

public:
  Input(bool reverse, const thrust::host_vector<OffsetT> &h_segment_sizes)
      : d_segment_sizes(h_segment_sizes)
      , d_offsets(d_segment_sizes.size() + 1)
      , h_offsets(d_segment_sizes.size() + 1)
      , reverse(reverse)
      , num_items(static_cast<OffsetT>(
          thrust::reduce(d_segment_sizes.begin(), d_segment_sizes.end())))
      , d_keys(num_items)
      , d_values(num_items)
  {
    update();
  }

  Input(thrust::host_vector<OffsetT> &h_offsets)
    : d_offsets(h_offsets)
    , h_offsets(h_offsets)
    , reverse(false)
    , num_items(h_offsets.back())
    , d_keys(num_items)
    , d_values(num_items)
  {
  }

  void shuffle()
  {
    thrust::shuffle(d_segment_sizes.begin(), d_segment_sizes.end(), random_engine);

    update();
  }

  OffsetT get_num_items() const
  {
    return num_items;
  }

  unsigned int get_num_segments() const
  {
    return static_cast<unsigned int>(d_segment_sizes.size());
  }

  const KeyT *get_d_keys() const
  {
    return thrust::raw_pointer_cast(d_keys.data());
  }

  thrust::device_vector<KeyT> &get_d_keys_vec()
  {
    return d_keys;
  }

  thrust::device_vector<MaskedValueT> &get_d_values_vec()
  {
    return d_values;
  }

  KeyT *get_d_keys()
  {
    return thrust::raw_pointer_cast(d_keys.data());
  }

  const thrust::host_vector<OffsetT>& get_h_offsets()
  {
    return h_offsets;
  }

  MaskedValueT *get_d_values()
  {
    return thrust::raw_pointer_cast(d_values.data());
  }

  const OffsetT *get_d_offsets() const
  {
    return thrust::raw_pointer_cast(d_offsets.data());
  }

  template <typename T>
  bool check_output_implementation(const T *keys_output)
  {
    thrust::device_vector<bool> is_segment_sorted(get_num_segments(), true);

    if (reverse)
    {
      thrust::transform(
        thrust::counting_iterator<unsigned int>(0),
                        thrust::counting_iterator<unsigned int>(
                          static_cast<unsigned int>(get_num_segments())),
                        is_segment_sorted.begin(),
                        DescendingSegmentChecker<T, OffsetT>(keys_output,
                                                             get_d_offsets()));
    }
    else
    {
      thrust::transform(
        thrust::counting_iterator<unsigned int>(0),
                        thrust::counting_iterator<unsigned int>(
                          static_cast<unsigned int>(get_num_segments())),
                        is_segment_sorted.begin(),
                        SegmentChecker<T, OffsetT>(keys_output,
                                                   get_d_offsets()));
    }

    return thrust::reduce(is_segment_sorted.begin(),
                          is_segment_sorted.end(),
                          true,
                          thrust::logical_and<bool>());
  }

  bool check_output(const KeyT *keys_output,
                    const MaskedValueT *values_output = nullptr)
  {
    const bool keys_ok = check_output_implementation(keys_output);
    const bool values_ok = (std::is_same<ValueT, hipcub::NullType>::value ||
                            values_output == nullptr)
                             ? true
                             : check_output_implementation(values_output);

    return keys_ok && values_ok;
  }

private:
  void update()
  {
    fill_offsets();
    gen_keys();
  }

  void fill_offsets()
  {
    thrust::copy(d_segment_sizes.begin(), d_segment_sizes.end(), d_offsets.begin());
    thrust::exclusive_scan(d_offsets.begin(), d_offsets.end(), d_offsets.begin(), 0u);
    thrust::copy(d_offsets.begin(), d_offsets.end(), h_offsets.begin());
  }

  void gen_keys()
  {
    const unsigned int total_segments =
      static_cast<unsigned int>(get_num_segments());

    if (reverse)
    {
      thrust::for_each(thrust::counting_iterator<unsigned int>(0),
                       thrust::counting_iterator<unsigned int>(total_segments),
                       IOTA<KeyT, OffsetT>(get_d_keys(),
                                           get_d_offsets()));
    }
    else
    {
      thrust::for_each(thrust::counting_iterator<unsigned int>(0),
                       thrust::counting_iterator<unsigned int>(total_segments),
                       ReversedIOTA<KeyT, OffsetT>(get_d_keys(),
                                                   get_d_offsets()));
    }

    thrust::copy(d_keys.begin(), d_keys.end(), d_values.begin());
  }
};

template <typename KeyT, typename OffsetT>
class InputDescription
{
  thrust::host_vector<OffsetT> segment_sizes;

public:
  InputDescription& add(const SizeGroupDescription &group)
  {
    if (static_cast<std::size_t>(group.segment_size) <
        static_cast<std::size_t>(std::numeric_limits<KeyT>::max()))
    {
      for (int i = 0; i < group.segments; i++)
      {
        segment_sizes.push_back(group.segment_size);
      }
    }

    return *this;
  }

  template <typename ValueT = hipcub::NullType>
  Input<KeyT, OffsetT, ValueT> gen(bool reverse)
  {
    return Input<KeyT, OffsetT, ValueT>(reverse, segment_sizes);
  }
};


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void Sort(bool pairs,
          bool descending,
          bool double_buffer,

          void *tmp_storage,
          std::size_t &temp_storage_bytes,

          KeyT *input_keys,
          KeyT *output_keys,

          ValueT *input_values,
          ValueT *output_values,

          OffsetT num_items,
          unsigned int num_segments,
          const OffsetT *d_offsets,

          int *keys_selector = nullptr,
          int *values_selector = nullptr)
{
  if (pairs)
  {
    if (descending)
    {
      if (double_buffer)
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(input_keys, output_keys);
        keys_buffer.selector = *keys_selector;

        hipcub::DoubleBuffer<ValueT> values_buffer(input_values, output_values);
        values_buffer.selector = *values_selector;

        CubDebugExit(
          hipcub::DeviceSegmentedSort::SortPairsDescending(tmp_storage,
                                                        temp_storage_bytes,
                                                        keys_buffer,
                                                        values_buffer,
                                                        num_items,
                                                        num_segments,
                                                        d_offsets,
                                                        d_offsets + 1,
                                                        0,
                                                        true));

        *keys_selector = keys_buffer.selector;
        *values_selector = values_buffer.selector;
      }
      else
      {
        CubDebugExit(
          hipcub::DeviceSegmentedSort::SortPairsDescending(tmp_storage,
                                                        temp_storage_bytes,
                                                        input_keys,
                                                        output_keys,
                                                        input_values,
                                                        output_values,
                                                        num_items,
                                                        num_segments,
                                                        d_offsets,
                                                        d_offsets + 1,
                                                        0,
                                                        true));
      }
    }
    else
    {
      if (double_buffer)
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(input_keys, output_keys);
        keys_buffer.selector = *keys_selector;

        hipcub::DoubleBuffer<ValueT> values_buffer(input_values, output_values);
        values_buffer.selector = *values_selector;

        CubDebugExit(hipcub::DeviceSegmentedSort::SortPairs(tmp_storage,
                                                         temp_storage_bytes,
                                                         keys_buffer,
                                                         values_buffer,
                                                         num_items,
                                                         num_segments,
                                                         d_offsets,
                                                         d_offsets + 1,
                                                         0,
                                                         true));

        *keys_selector = keys_buffer.selector;
        *values_selector = values_buffer.selector;
      }
      else
      {
        CubDebugExit(hipcub::DeviceSegmentedSort::SortPairs(tmp_storage,
                                                         temp_storage_bytes,
                                                         input_keys,
                                                         output_keys,
                                                         input_values,
                                                         output_values,
                                                         num_items,
                                                         num_segments,
                                                         d_offsets,
                                                         d_offsets + 1,
                                                         0,
                                                         true));
      }
    }
  }
  else
  {
    if (descending)
    {
      if (double_buffer)
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(input_keys, output_keys);
        keys_buffer.selector = *keys_selector;

        CubDebugExit(
          hipcub::DeviceSegmentedSort::SortKeysDescending(tmp_storage,
                                                       temp_storage_bytes,
                                                       keys_buffer,
                                                       num_items,
                                                       num_segments,
                                                       d_offsets,
                                                       d_offsets + 1,
                                                       0,
                                                       true));

        *keys_selector = keys_buffer.selector;
      }
      else
      {
        CubDebugExit(
          hipcub::DeviceSegmentedSort::SortKeysDescending(tmp_storage,
                                                       temp_storage_bytes,
                                                       input_keys,
                                                       output_keys,
                                                       num_items,
                                                       num_segments,
                                                       d_offsets,
                                                       d_offsets + 1,
                                                       0,
                                                       true));
      }
    }
    else
    {
      if (double_buffer)
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(input_keys, output_keys);
        keys_buffer.selector = *keys_selector;

        CubDebugExit(hipcub::DeviceSegmentedSort::SortKeys(tmp_storage,
                                                        temp_storage_bytes,
                                                        keys_buffer,
                                                        num_items,
                                                        num_segments,
                                                        d_offsets,
                                                        d_offsets + 1,
                                                        0,
                                                        true));

        *keys_selector = keys_buffer.selector;
      }
      else
      {
        CubDebugExit(hipcub::DeviceSegmentedSort::SortKeys(tmp_storage,
                                                        temp_storage_bytes,
                                                        input_keys,
                                                        output_keys,
                                                        num_items,
                                                        num_segments,
                                                        d_offsets,
                                                        d_offsets + 1,
                                                        0,
                                                        true));
      }
    }
  }
}

template <typename KeyT,
          typename ValueT,
          typename OffsetT>
std::size_t Sort(bool pairs,
                 bool descending,
                 bool double_buffer,

                 KeyT *input_keys,
                 KeyT *output_keys,

                 ValueT *input_values,
                 ValueT *output_values,

                 OffsetT num_items,
                 unsigned int num_segments,
                 const OffsetT *d_offsets,

                 int *keys_selector   = nullptr,
                 int *values_selector = nullptr)
{
  std::size_t temp_storage_bytes = 42ul;

  Sort<KeyT, ValueT, OffsetT>(pairs,
                              descending,
                              double_buffer,
                              nullptr,
                              temp_storage_bytes,
                              input_keys,
                              output_keys,
                              input_values,
                              output_values,
                              num_items,
                              num_segments,
                              d_offsets,
                              keys_selector,
                              values_selector);

  thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
  std::uint8_t *d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  Sort<KeyT, ValueT, OffsetT>(pairs,
                              descending,
                              double_buffer,
                              d_temp_storage,
                              temp_storage_bytes,
                              input_keys,
                              output_keys,
                              input_values,
                              output_values,
                              num_items,
                              num_segments,
                              d_offsets,
                              keys_selector,
                              values_selector);

  return temp_storage_bytes;
}


constexpr bool keys = false;
constexpr bool pairs = true;

constexpr bool ascending = false;
constexpr bool descending = true;

constexpr bool pointers = false;
constexpr bool double_buffer = true;


void TestZeroSegments()
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;
  using ValueT = std::uint64_t;
  using OffsetT = std::uint32_t;

  for (bool sort_keys: { keys, pairs })
  {
    for (bool sort_ascending: { ascending, descending })
    {
      for (bool sort_pointers: { pointers, double_buffer })
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
        hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
        values_buffer.selector = 1;

        const std::size_t temp_storage_bytes =
          Sort<KeyT, ValueT, OffsetT>(sort_keys,
                                      sort_ascending,
                                      sort_pointers,
                                      nullptr,
                                      nullptr,
                                      nullptr,
                                      nullptr,
                                      OffsetT{},
                                      OffsetT{},
                                      nullptr,
                                      &keys_buffer.selector,
                                      &values_buffer.selector);

        AssertEquals(keys_buffer.selector, 0);
        AssertEquals(values_buffer.selector, 1);
        AssertEquals(temp_storage_bytes, 0ul);
      }
    }
  }
}


void TestEmptySegments(unsigned int segments)
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;
  using ValueT = std::uint64_t;
  using OffsetT = std::uint32_t;

  thrust::device_vector<OffsetT> offsets(segments + 1, OffsetT{});
  const OffsetT *d_offsets = thrust::raw_pointer_cast(offsets.data());

  for (bool sort_keys: { keys, pairs })
  {
    for (bool sort_ascending: { ascending, descending })
    {
      for (bool sort_pointers: { pointers, double_buffer })
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
        hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
        values_buffer.selector = 1;

        const std::size_t temp_storage_bytes =
          Sort<KeyT, ValueT, OffsetT>(sort_keys,
                                      sort_ascending,
                                      sort_pointers,
                                      nullptr,
                                      nullptr,
                                      nullptr,
                                      nullptr,
                                      OffsetT{},
                                      segments,
                                      d_offsets,
                                      &keys_buffer.selector,
                                      &values_buffer.selector);

        AssertEquals(keys_buffer.selector, 0);
        AssertEquals(values_buffer.selector, 1);
        AssertEquals(temp_storage_bytes, 0ul);
      }
    }
  }
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void TestSameSizeSegments(OffsetT segment_size,
                          unsigned int segments,
                          bool skip_values = false)
{
  const OffsetT num_items = segment_size * segments;

  thrust::device_vector<OffsetT> offsets(segments + 1);
  thrust::sequence(offsets.begin(),
                   offsets.end(),
                   OffsetT{},
                   OffsetT{segment_size});

  const OffsetT *d_offsets = thrust::raw_pointer_cast(offsets.data());

  const KeyT target_key = KeyT{42};
  const ValueT target_value = ValueT{42};

  thrust::device_vector<KeyT> keys_input(num_items);
  thrust::device_vector<KeyT> keys_output(num_items);

  KeyT *d_keys_input  = thrust::raw_pointer_cast(keys_input.data());
  KeyT *d_keys_output = thrust::raw_pointer_cast(keys_output.data());

  thrust::device_vector<ValueT> values_input(num_items);
  thrust::device_vector<ValueT> values_output(num_items);

  ValueT *d_values_input  = thrust::raw_pointer_cast(values_input.data());
  ValueT *d_values_output = thrust::raw_pointer_cast(values_output.data());

  for (bool sort_pairs: { keys, pairs })
  {
    if (sort_pairs)
    {
      if (skip_values)
      {
        continue;
      }
    }

    for (bool sort_descending: { ascending, descending })
    {
      for (bool sort_buffers: { pointers, double_buffer })
      {
        hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
        hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
        values_buffer.selector = 1;

        thrust::fill(keys_input.begin(), keys_input.end(), target_key);
        thrust::fill(keys_output.begin(), keys_output.end(), KeyT{});

        if (sort_pairs)
        {
          if (sort_buffers)
          {
            thrust::fill(values_input.begin(), values_input.end(), ValueT{});
            thrust::fill(values_output.begin(), values_output.end(), target_value);
          }
          else
          {
            thrust::fill(values_input.begin(), values_input.end(), target_value);
            thrust::fill(values_output.begin(), values_output.end(), ValueT{});
          }
        }

        const std::size_t temp_storage_bytes =
          Sort<KeyT, ValueT, OffsetT>(sort_pairs,
                                      sort_descending,
                                      sort_buffers,
                                      d_keys_input,
                                      d_keys_output,
                                      d_values_input,
                                      d_values_output,
                                      num_items,
                                      segments,
                                      d_offsets,
                                      &keys_buffer.selector,
                                      &values_buffer.selector);

        // If temporary storage size is defined by extra keys storage
        if (sort_buffers)
        {
          if (2 * segments * sizeof(unsigned int) < num_items * sizeof(KeyT))
          {
            std::size_t extra_temp_storage_bytes{};

            Sort(sort_pairs,
                 sort_descending,
                 pointers,
                 nullptr,
                 extra_temp_storage_bytes,
                 d_keys_input,
                 d_keys_output,
                 d_values_input,
                 d_values_output,
                 num_items,
                 segments,
                 d_offsets,
                 &keys_buffer.selector,
                 &values_buffer.selector);

            AssertTrue(extra_temp_storage_bytes > temp_storage_bytes);
          }
        }

        {
          const std::size_t items_selected =
            keys_buffer.selector || !sort_buffers
              ? thrust::count(keys_output.begin(),
                              keys_output.end(),
                              target_key)
              : thrust::count(keys_input.begin(), keys_input.end(), target_key);
          AssertEquals(items_selected, num_items);
        }

        if (sort_pairs)
        {
          const std::size_t items_selected = [&]() -> std::size_t {
            if (sort_buffers)
            {
              return values_buffer.selector
                       ? thrust::count(values_output.begin(),
                                       values_output.end(),
                                       target_value)
                       : thrust::count(values_input.begin(),
                                       values_input.end(),
                                       target_value);
            }

            return thrust::count(values_output.begin(),
                                 values_output.end(),
                                 target_value);
          } ();

          AssertEquals(items_selected, num_items);
        }
      }
    }
  }
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void InputTest(bool sort_descending,
               Input<KeyT, OffsetT, ValueT> &input)
{
  thrust::device_vector<KeyT> keys_output(input.get_num_items());
  KeyT *d_keys_output = thrust::raw_pointer_cast(keys_output.data());

  thrust::device_vector<ValueT> values_output(input.get_num_items());
  ValueT *d_values_output = thrust::raw_pointer_cast(values_output.data());

  for (bool sort_pairs: { keys, pairs })
  {
    for (bool sort_buffers: { pointers, double_buffer })
    {
      for (int iteration = 0; iteration < MAX_ITERATIONS; iteration++)
      {
        thrust::fill(keys_output.begin(), keys_output.end(), KeyT{});
        thrust::fill(values_output.begin(), values_output.end(), ValueT{});

        hipcub::DoubleBuffer<KeyT> keys_buffer(input.get_d_keys(), d_keys_output);
        hipcub::DoubleBuffer<ValueT> values_buffer(input.get_d_values(), d_values_output);

        Sort<KeyT, ValueT, OffsetT>(
          sort_pairs,
          sort_descending,
          sort_buffers,
          input.get_d_keys(),
          d_keys_output,
          input.get_d_values(),
          d_values_output,
          input.get_num_items(),
          input.get_num_segments(),
          input.get_d_offsets(),
          &keys_buffer.selector,
          &values_buffer.selector);

        if (sort_buffers)
        {
          if (sort_pairs)
          {
            AssertTrue(input.check_output(keys_buffer.Current(),
                                          values_buffer.Current()));
          }
          else
          {
            AssertTrue(input.check_output(keys_buffer.Current()));
          }
        }
        else
        {
          if (sort_pairs)
          {
            AssertTrue(input.check_output(d_keys_output, d_values_output));
          }
          else
          {
            AssertTrue(input.check_output(d_keys_output));
          }
        }

        input.shuffle();
      }
    }
  }
}


template <typename T,
          typename OffsetT>
bool compare_two_outputs(const thrust::host_vector<OffsetT> &offsets,
                         const thrust::host_vector<T> &lhs,
                         const thrust::host_vector<T> &rhs)
{
  const auto num_segments = static_cast<unsigned int>(offsets.size() - 1);

  for (std::size_t segment_id = 0; segment_id < num_segments; segment_id++)
  {
    auto lhs_begin = lhs.cbegin() + offsets[segment_id];
    auto lhs_end = lhs.cbegin() + offsets[segment_id + 1];
    auto rhs_begin = rhs.cbegin() + offsets[segment_id];

    auto err = thrust::mismatch(lhs_begin, lhs_end, rhs_begin);

    if (err.first != lhs_end)
    {
      const auto idx = thrust::distance(lhs_begin, err.first);
      const auto segment_size = std::distance(lhs_begin, lhs_end);

      std::cerr << "Mismatch in segment " << segment_id
                << " at position " << idx << " / " << segment_size
                << ": "
                << static_cast<std::uint64_t>(lhs_begin[idx]) << " vs "
                << static_cast<std::uint64_t>(rhs_begin[idx]) << " ("
                << typeid(lhs_begin[idx]).name() << ")" << std::endl;

      return false;
    }
  }

  return true;
}


template <typename KeyT,
          typename ValueT>
void RandomizeInput(thrust::host_vector<KeyT> &h_keys,
                    thrust::host_vector<ValueT> &h_values)
{
  for (std::size_t i = 0; i < h_keys.size(); i++)
  {
    h_keys[i] = RandomValue(std::numeric_limits<KeyT>::max());
    h_values[i] = RandomValue(std::numeric_limits<ValueT>::max());
  }
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void HostReferenceSort(bool sort_pairs,
                       bool sort_descending,
                       unsigned int num_segments,
                       const thrust::host_vector<OffsetT> &h_offsets,
                       thrust::host_vector<KeyT> &h_keys,
                       thrust::host_vector<ValueT> &h_values)
{
  for (unsigned int segment_i = 0;
       segment_i < num_segments;
       segment_i++)
  {
    const OffsetT segment_begin = h_offsets[segment_i];
    const OffsetT segment_end   = h_offsets[segment_i + 1];

    if (sort_pairs)
    {
      if (sort_descending)
      {
        thrust::sort_by_key(h_keys.begin() + segment_begin,
                            h_keys.begin() + segment_end,
                            h_values.begin() + segment_begin,
                            thrust::greater<KeyT>{});
      }
      else
      {
        thrust::sort_by_key(h_keys.begin() + segment_begin,
                            h_keys.begin() + segment_end,
                            h_values.begin() + segment_begin);
      }
    }
    else
    {
      if (sort_descending)
      {
        thrust::sort(h_keys.begin() + segment_begin,
                     h_keys.begin() + segment_end,
                     thrust::greater<KeyT>{});
      }
      else
      {
        thrust::sort(h_keys.begin() + segment_begin,
                     h_keys.begin() + segment_end);
      }
    }
  }
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void DumpInput(bool sort_pairs,
               bool sort_descending,
               bool sort_buffers,
               Input<KeyT, OffsetT, ValueT> &input,
               thrust::host_vector<KeyT> &h_keys,
               thrust::host_vector<ValueT> &h_values)
{
  const thrust::host_vector<OffsetT> &h_offsets = input.get_h_offsets();

  std::cout << "sort pairs: " << sort_pairs << "\n";
  std::cout << "sort descending: " << sort_descending << "\n";
  std::cout << "sort buffers: " << sort_buffers << "\n";
  std::cout << "num_items: " << input.get_num_items() << "\n";
  std::cout << "num_segments: " << input.get_num_segments() << "\n";
  std::cout << "key type: " << typeid(h_keys[0]).name() << "\n";
  std::cout << "value type: " << typeid(h_values[0]).name() << "\n";
  std::cout << "offset type: " << typeid(h_offsets[0]).name() << "\n";

  std::ofstream offsets_dump("offsets", std::ios::binary);
  offsets_dump.write(reinterpret_cast<const char *>(
                       thrust::raw_pointer_cast(h_offsets.data())),
                     sizeof(OffsetT) * h_offsets.size());

  std::ofstream keys_dump("keys", std::ios::binary);
  keys_dump.write(reinterpret_cast<const char *>(
                    thrust::raw_pointer_cast(h_keys.data())),
                  sizeof(KeyT) * h_keys.size());

  std::ofstream values_dump("values", std::ios::binary);
  values_dump.write(reinterpret_cast<const char *>(
                      thrust::raw_pointer_cast(h_values.data())),
                    sizeof(ValueT) * h_values.size());
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void InputTestRandom(Input<KeyT, OffsetT, ValueT> &input)
{
  thrust::host_vector<KeyT> h_keys_output(input.get_num_items());
  thrust::device_vector<KeyT> keys_output(input.get_num_items());

  thrust::host_vector<ValueT> h_values_output(input.get_num_items());
  thrust::device_vector<ValueT> values_output(input.get_num_items());

  KeyT *d_keys_output = thrust::raw_pointer_cast(keys_output.data());
  ValueT *d_values_output = thrust::raw_pointer_cast(values_output.data());

  thrust::host_vector<KeyT> h_keys(input.get_num_items());
  thrust::host_vector<ValueT> h_values(input.get_num_items());

  const thrust::host_vector<OffsetT> &h_offsets = input.get_h_offsets();

  for (bool sort_pairs: { keys, pairs })
  {
    for (bool sort_descending: { ascending, descending })
    {
      for (bool sort_buffers: { pointers, double_buffer })
      {
        for (int iteration = 0; iteration < MAX_ITERATIONS / 10; iteration++)
        {
          RandomizeInput(h_keys, h_values);

#if STORE_ON_FAILURE
          auto h_keys_backup = h_keys;
          auto h_values_backup = h_values;
#endif

          input.get_d_keys_vec()   = h_keys;
          input.get_d_values_vec() = h_values;

          hipcub::DoubleBuffer<KeyT> keys_buffer(input.get_d_keys(), d_keys_output);
          hipcub::DoubleBuffer<ValueT> values_buffer(input.get_d_values(), d_values_output);

          Sort<KeyT, ValueT, OffsetT>(
            sort_pairs,
            sort_descending,
            sort_buffers,
            input.get_d_keys(),
            d_keys_output,
            input.get_d_values(),
            d_values_output,
            input.get_num_items(),
            input.get_num_segments(),
            input.get_d_offsets(),
            &keys_buffer.selector,
            &values_buffer.selector);

          HostReferenceSort(sort_pairs,
                            sort_descending,
                            input.get_num_segments(),
                            h_offsets,
                            h_keys,
                            h_values);

          if (sort_buffers)
          {
            if (keys_buffer.selector)
            {
              h_keys_output = keys_output;
            }
            else
            {
              h_keys_output = input.get_d_keys_vec();
            }

            if (values_buffer.selector)
            {
              h_values_output = values_output;
            }
            else
            {
              h_values_output = input.get_d_values_vec();
            }
          }
          else
          {
            h_keys_output = keys_output;
            h_values_output = values_output;
          }

          const bool keys_ok =
            compare_two_outputs(h_offsets, h_keys, h_keys_output);

          const bool values_ok =
            sort_pairs
              ? compare_two_outputs(h_offsets, h_values, h_values_output)
              : true;

#if STORE_ON_FAILURE
          if (!keys_ok || !values_ok)
          {
            DumpInput<KeyT, ValueT, OffsetT>(sort_pairs,
                                             sort_descending,
                                             sort_buffers,
                                             input,
                                             h_keys_backup,
                                             h_values_backup);
          }
#endif

          AssertTrue(keys_ok);
          AssertTrue(values_ok);

          input.shuffle();
        }
      }
    }
  }
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void EdgePatternsTest()
{
  for (bool sort_descending: { ascending, descending })
  {
    Input<KeyT, OffsetT, ValueT> edge_cases =
      InputDescription<KeyT, OffsetT>()
        .add({420, 0})
        .add({420, 1})
        .add({420, 2})
        .add({420, 8})
        .add({420, 9})
        .add({420, 10})
        .add({420, 17})
        .add({42, 18})
        .add({42, 19})
        .add({42, 26})
        .add({42, 27})
        .add({42, 28})
        .add({42, 35})
        .add({42, 36})
        .add({42, 37})
        .add({42, 286})
        .add({42, 287})
        .add({42, 288})
        .add({42, 5887})
        .add({42, 5888})
        .add({42, 5889})
        .add({2, 23552})
        .template gen<ValueT>(sort_descending);

    InputTest<KeyT, ValueT, OffsetT>(sort_descending, edge_cases);
  }
}

template <typename KeyT,
          typename ValueT,
          typename OffsetT>
Input<KeyT, OffsetT, ValueT> GenRandomInput(OffsetT max_items,
                                            OffsetT min_segments,
                                            OffsetT max_segments,
                                            bool descending)
{
  std::size_t items_generated {};
  const std::size_t segments_num = RandomValue(max_segments) + min_segments;

  thrust::host_vector<OffsetT> segment_sizes;
  segment_sizes.reserve(segments_num);

  const OffsetT max_segment_size = 6000;

  for (std::size_t segment_id = 0; segment_id < segments_num; segment_id++)
  {
    const OffsetT segment_size_raw = RandomValue(max_segment_size);
    const OffsetT segment_size = segment_size_raw > OffsetT{0} ? segment_size_raw
                                                               : OffsetT{0};

    if (segment_size + items_generated > max_items)
    {
      break;
    }

    items_generated += segment_size;
    segment_sizes.push_back(segment_size);
  }

  return Input<KeyT, OffsetT, ValueT>{descending, segment_sizes};
}

template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void RandomTest(OffsetT min_segments,
                OffsetT max_segments)
{
  const OffsetT max_items = 10000000;

  for (int iteration = 0; iteration < 10 * MAX_ITERATIONS; iteration++)
  {
    Input<KeyT, OffsetT, ValueT> edge_cases =
      GenRandomInput<KeyT, ValueT, OffsetT>(max_items,
                                            min_segments,
                                            max_segments,
                                            descending);

    InputTestRandom(edge_cases);
  }
}

template <typename KeyT,
          typename OffsetT>
void TestKeys()
{
  const bool skip_values = true;

  for (OffsetT segment_size : {1, 1024, 24 * 1024})
  {
    for (int segments : {1, 1024})
    {
      TestSameSizeSegments<KeyT, KeyT, OffsetT>(segment_size,
                                                segments,
                                                skip_values);
    }
  }
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void TestPairs()
{
  for (OffsetT segment_size: { 1, 1024, 24 * 1024 })
  {
    for (int segments: { 1, 1024 })
    {
      TestSameSizeSegments<KeyT, ValueT, OffsetT>(segment_size, segments);
    }
  }

  RandomTest<KeyT, ValueT, OffsetT>(1 << 2, 1 << 8);
  RandomTest<KeyT, ValueT, OffsetT>(1 << 9, 1 << 19);
}

template <typename T,
          typename OffsetT>
void TestKeysAndPairs()
{
  TestKeys<T, OffsetT>();
  TestPairs<T, T, OffsetT>();
}


template <typename KeyT,
          typename ValueT,
          typename OffsetT>
void InputTestFromFiles()
{
  (void)MAX_ITERATIONS;

  const bool sort_pairs      = true;
  const bool sort_descending = true;
  const bool sort_buffers    = false;

  const OffsetT num_items    = 9998991;
  const unsigned int num_segments = 3298;

  thrust::host_vector<OffsetT> h_offsets(num_segments + 1);
  std::ifstream offsets_file("offsets", std::ios::binary);
  offsets_file.read(reinterpret_cast<char *>(
                      thrust::raw_pointer_cast(h_offsets.data())),
                    sizeof(OffsetT) * (num_segments + 1));
  thrust::device_vector<OffsetT> d_offsets = h_offsets;

  thrust::host_vector<KeyT> h_keys(num_items);
  std::ifstream keys_file("keys", std::ios::binary);
  keys_file.read(reinterpret_cast<char *>(
                   thrust::raw_pointer_cast(h_keys.data())),
                 sizeof(KeyT) * num_items);

  thrust::host_vector<ValueT> h_values(num_items);
  std::ifstream values_file("values", std::ios::binary);
  values_file.read(reinterpret_cast<char *>(
                     thrust::raw_pointer_cast(h_values.data())),
                   sizeof(ValueT) * num_items);

  thrust::host_vector<KeyT> h_keys_output(num_items);
  thrust::device_vector<KeyT> keys_output(num_items);

  thrust::host_vector<ValueT> h_values_output(num_items);
  thrust::device_vector<ValueT> values_output(num_items);

  thrust::device_vector<KeyT> d_keys = h_keys;
  thrust::device_vector<ValueT> d_values = h_values;

  KeyT *d_keys_input = thrust::raw_pointer_cast(d_keys.data());
  ValueT *d_values_input = thrust::raw_pointer_cast(d_values.data());
  KeyT *d_keys_output     = thrust::raw_pointer_cast(keys_output.data());
  ValueT *d_values_output = thrust::raw_pointer_cast(values_output.data());

  hipcub::DoubleBuffer<KeyT> keys_buffer(d_keys_input, d_keys_output);
  hipcub::DoubleBuffer<ValueT> values_buffer(d_values_input, d_values_output);

  Sort<KeyT, ValueT, OffsetT>(sort_pairs,
                              sort_descending,
                              sort_buffers,
                              d_keys_input,
                              d_keys_output,
                              d_values_input,
                              d_values_output,
                              num_items,
                              num_segments,
                              thrust::raw_pointer_cast(d_offsets.data()),
                              &keys_buffer.selector,
                              &values_buffer.selector);

  HostReferenceSort(sort_pairs,
                    sort_descending,
                    num_segments,
                    h_offsets,
                    h_keys,
                    h_values);

  if (sort_buffers)
  {
    if (keys_buffer.selector)
    {
      h_keys_output = keys_output;
    }
    else
    {
      h_keys_output = d_keys;
    }

    if (values_buffer.selector)
    {
      h_values_output = values_output;
    }
    else
    {
      h_values_output = d_values;
    }
  }
  else
  {
    h_keys_output   = keys_output;
    h_values_output = values_output;
  }

  const bool keys_ok = compare_two_outputs(h_offsets, h_keys, h_keys_output);

  const bool values_ok =
    sort_pairs ? compare_two_outputs(h_offsets, h_values, h_values_output)
               : true;

  AssertTrue(keys_ok);
  AssertTrue(values_ok);
}

int main(int argc, char** argv)
{
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  TestZeroSegments();
  TestEmptySegments(1 << 2);
  TestEmptySegments(1 << 22);

  TestKeysAndPairs<std::uint8_t,  std::uint32_t>();
  TestKeysAndPairs<std::uint16_t, std::uint32_t>();
  TestKeysAndPairs<std::uint32_t, std::uint32_t>();
  TestKeysAndPairs<std::uint64_t, std::uint32_t>();
  TestKeysAndPairs<std::uint64_t, std::uint64_t>();
  TestPairs<std::uint8_t, std::uint64_t, std::uint32_t>();
  TestPairs<std::int64_t, std::uint64_t, std::uint32_t>();

  return 0;
}
