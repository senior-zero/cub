/******************************************************************************
* Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright
*       notice, this list of conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright
*       notice, this list of conditions and the following disclaimer in the
*       documentation and/or other materials provided with the distribution.
*     * Neither the name of the NVIDIA CORPORATION nor the
*       names of its contributors may be used to endorse or promote products
*       derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
* ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*
******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/device/device_for.cuh>

#include "test_util.h"


struct PrintT
{
  __device__ void operator()(int i) const
  {
    printf("%d\n", i);
  }
};


int main(int argc, char** argv)
{
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  auto tuning = hipcub::TuneForEach<hipcub::ForEachAlgorithm::STRIPED>(
    hipcub::ForEachConfigurationSpace{}.Add<1024, 2>()
                                    .Add<256, 1>());

  hipcub::DeviceFor::Bulk(4, PrintT{}, 0, true, tuning);
}
