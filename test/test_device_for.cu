#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/device/device_for.cuh>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/random.h>

#include "test_util.h"

template <typename OffsetT>
struct Incrementer
{
  int *d_counts{};

  __device__ void operator()(OffsetT i)
  {
    // Check if some `i` were served more than once
    atomicAdd(d_counts + static_cast<int>(i), 1);
  }
};

template <typename OffsetT>
class OffsetProxy
{
  OffsetT m_offset;

public:
  __host__ __device__ OffsetProxy(OffsetT offset)
      : m_offset(offset)
  {}

  __host__ __device__ operator OffsetT() const { return m_offset; }
};

struct ItemOverwriter
{
  const std::size_t *d_input;
  const std::size_t magic_value;

  __device__ void operator()(const std::size_t &i) const
  {
    if (i == magic_value)
    {
      const std::size_t *d_ptr     = &i;
      const std::size_t offset     = static_cast<std::size_t>(d_ptr - d_input);
      const_cast<std::size_t &>(i) = offset;
    }
  }
};

template <typename OffsetT>
void TestBulkDefault(OffsetT num_items)
{
  thrust::device_vector<int> counts(num_items);
  int *d_counts = thrust::raw_pointer_cast(counts.data());

  hipcub::DeviceFor::Bulk(num_items, Incrementer<OffsetT>{d_counts}, {}, true);

  const OffsetT num_of_once_marked_items =
    static_cast<OffsetT>(thrust::count(counts.begin(), counts.end(), 1));

  AssertEquals(num_items, num_of_once_marked_items);
}

template <typename OffsetT,
          hipcub::ForEachAlgorithm Algorithm,
          unsigned BlockThreads,
          unsigned ItemsPerThread>
void TestBulkTuned(OffsetT num_items)
{
  auto tuning = hipcub::TuneForEach<Algorithm>(
    hipcub::ForEachConfigurationSpace{}.Add<BlockThreads, ItemsPerThread>());

  thrust::device_vector<int> counts(num_items);
  int *d_counts = thrust::raw_pointer_cast(counts.data());

  hipcub::DeviceFor::Bulk(num_items,
                       Incrementer<OffsetT>{d_counts},
                       {},
                       true,
                       tuning);

  const OffsetT num_of_once_marked_items =
    static_cast<OffsetT>(thrust::count(counts.begin(), counts.end(), 1));

  AssertEquals(num_items, num_of_once_marked_items);
}

template <typename OffsetT>
void TestBulkTuned(OffsetT num_items)
{
  constexpr auto block_striped = hipcub::ForEachAlgorithm::BLOCK_STRIPED;

  TestBulkTuned<OffsetT, block_striped, 32, 28>(num_items);
  TestBulkTuned<OffsetT, block_striped, 128, 8>(num_items);
  TestBulkTuned<OffsetT, block_striped, 256, 2>(num_items);
  TestBulkTuned<OffsetT, block_striped, 512, 3>(num_items);
  TestBulkTuned<OffsetT, block_striped, 1024, 1>(num_items);
}

template <typename OffsetT>
void TestBulk(OffsetT num_items)
{
  TestBulkDefault<OffsetT>(num_items);
  TestBulkTuned<OffsetT>(num_items);
}

template <typename OffsetT>
void TestBulkRandom(thrust::default_random_engine &rng)
{
  const int num_iterations = 8;
  const OffsetT max_items  = 2 << 26; // Up to 512 MB
  thrust::uniform_int_distribution<OffsetT> dist(0, max_items);

  for (int iteration = 0; iteration < num_iterations; iteration++)
  {
    OffsetT num_items = dist(rng);
    TestBulk<OffsetT>(num_items);
  }
}

template <typename OffsetT>
void TestBulkEdgeCases()
{
  TestBulk<OffsetT>(0);

  for (int power_of_two = 0; power_of_two < 26; power_of_two += 2)
  {
    TestBulk<OffsetT>(static_cast<OffsetT>(2 << power_of_two) - 1);
    TestBulk<OffsetT>(static_cast<OffsetT>(2 << power_of_two));
    TestBulk<OffsetT>(static_cast<OffsetT>(2 << power_of_two) + 1);
  }
}

template <typename OffsetT>
void TestBulk(thrust::default_random_engine &rng)
{
  TestBulkRandom<OffsetT>(rng);
  TestBulkEdgeCases<OffsetT>();
}

void TestBulk(thrust::default_random_engine &rng)
{
  TestBulk<int>(rng);
  TestBulk<std::size_t>(rng);
}

template <typename OffsetT>
void TestForEachDefault(OffsetT num_items)
{
  thrust::device_vector<int> counts(num_items);
  thrust::device_vector<OffsetProxy<OffsetT>> input(num_items, OffsetT{});
  thrust::sequence(input.begin(), input.end(), OffsetT{});

  int *d_counts                 = thrust::raw_pointer_cast(counts.data());
  OffsetProxy<OffsetT> *d_input = thrust::raw_pointer_cast(input.data());

  hipcub::DeviceFor::ForEachN(d_input,
                           num_items,
                           Incrementer<OffsetProxy<OffsetT>>{d_counts},
                           {},
                           true);

  const OffsetT num_of_once_marked_items =
    static_cast<OffsetT>(thrust::count(counts.begin(), counts.end(), 1));

  AssertEquals(num_items, num_of_once_marked_items);
}

template <typename OffsetT,
          hipcub::ForEachAlgorithm Algorithm,
          hipcub::CacheLoadModifier LoadModifier,
          unsigned BlockThreads,
          unsigned ItemsPerThread>
void TestForEachTuned(OffsetT num_items)
{
  auto tuning = hipcub::TuneForEach<Algorithm, LoadModifier>(
    hipcub::ForEachConfigurationSpace{}.Add<BlockThreads, ItemsPerThread>());

  thrust::device_vector<int> counts(num_items);
  thrust::device_vector<OffsetProxy<OffsetT>> input(num_items, OffsetT{});
  thrust::sequence(input.begin(), input.end(), OffsetT{});

  int *d_counts                 = thrust::raw_pointer_cast(counts.data());
  OffsetProxy<OffsetT> *d_input = thrust::raw_pointer_cast(input.data());

  hipcub::DeviceFor::ForEachN(d_input,
                           num_items,
                           Incrementer<OffsetProxy<OffsetT>>{d_counts},
                           {},
                           true,
                           tuning);

  const OffsetT num_of_once_marked_items =
    static_cast<OffsetT>(thrust::count(counts.begin(), counts.end(), 1));

  AssertEquals(num_items, num_of_once_marked_items);
}

template <typename OffsetT, hipcub::CacheLoadModifier LoadModifier>
void TestForEachTuned(OffsetT num_items)
{
  constexpr auto block_striped = hipcub::ForEachAlgorithm::BLOCK_STRIPED;

  TestForEachTuned<OffsetT, block_striped, LoadModifier, 32, 28>(num_items);
  TestForEachTuned<OffsetT, block_striped, LoadModifier, 128, 8>(num_items);
  TestForEachTuned<OffsetT, block_striped, LoadModifier, 256, 7>(num_items);
  TestForEachTuned<OffsetT, block_striped, LoadModifier, 512, 3>(num_items);
  TestForEachTuned<OffsetT, block_striped, LoadModifier, 1024, 1>(num_items);
}

template <typename OffsetT>
void TestForEachTuned(OffsetT num_items)
{
  TestForEachTuned<OffsetT, hipcub::CacheLoadModifier::LOAD_DEFAULT>(num_items);
  TestForEachTuned<OffsetT, hipcub::CacheLoadModifier::LOAD_CA>(num_items);
  TestForEachTuned<OffsetT, hipcub::CacheLoadModifier::LOAD_CS>(num_items);
}

template <typename OffsetT>
void TestForEach(OffsetT num_items)
{
  // TODO Return once BLOCK_STRIPED_VECTORIZED is tested
  // TestForEachDefault<OffsetT>(num_items);
  TestForEachTuned<OffsetT>(num_items);
}

template <typename OffsetT>
void TestForEachRandom(thrust::default_random_engine &rng)
{
  const int num_iterations = 8;
  const OffsetT max_items  = 2 << 26; // Up to 512 MB
  thrust::uniform_int_distribution<OffsetT> dist(0, max_items);

  for (int iteration = 0; iteration < num_iterations; iteration++)
  {
    OffsetT num_items = dist(rng);
    TestForEach<OffsetT>(num_items);
  }
}

template <typename OffsetT>
void TestForEachEdgeCases()
{
  TestForEach<OffsetT>(0);

  for (int power_of_two = 0; power_of_two < 26; power_of_two += 4)
  {
    TestForEach<OffsetT>(static_cast<OffsetT>(2 << power_of_two) - 1);
    TestForEach<OffsetT>(static_cast<OffsetT>(2 << power_of_two));
    TestForEach<OffsetT>(static_cast<OffsetT>(2 << power_of_two) + 1);
  }
}

template <typename OffsetT>
void TestForEach(thrust::default_random_engine &rng)
{
  TestForEachRandom<OffsetT>(rng);
  TestForEachEdgeCases<OffsetT>();
}

void TestForEachIterator()
{
  const int num_items = 42 * 1024;
  thrust::device_vector<int> counts(num_items);
  int *d_counts = thrust::raw_pointer_cast(counts.data());
  auto begin    = thrust::make_counting_iterator(0);

  hipcub::DeviceFor::ForEachN(begin,
                           num_items,
                           Incrementer<int>{d_counts},
                           {},
                           true);

  const int num_of_once_marked_items =
    static_cast<int>(thrust::count(counts.begin(), counts.end(), 1));

  AssertEquals(num_items, num_of_once_marked_items);
}

template <hipcub::CacheLoadModifier LoadModifier>
void TestForEachOverwrite()
{
  const std::size_t num_items   = 42;
  const std::size_t magic_value = num_items + 1; // expected in ItemOverwriter

  thrust::device_vector<std::size_t> input(num_items, magic_value);

  const std::size_t *d_input = thrust::raw_pointer_cast(input.data());

  // Load modifier can restrict the ability to take the address of an element
  auto tuning =
    hipcub::TuneForEach<hipcub::ForEachAlgorithm::BLOCK_STRIPED, LoadModifier>(
      hipcub::ForEachConfigurationSpace{}.Add<256, 2>());

  hipcub::DeviceFor::ForEachN(d_input,
                           num_items,
                           ItemOverwriter{d_input, magic_value},
                           {},
                           true, 
                           tuning);

  if (LoadModifier == hipcub::CacheLoadModifier::LOAD_DEFAULT)
  {
    AssertTrue(thrust::equal(input.begin(),
                             input.end(),
                             thrust::make_counting_iterator(std::size_t{})));
  }
  else
  {
    const std::size_t num_magic_values = static_cast<std::size_t>(
      thrust::count(input.begin(), input.end(), magic_value));

    AssertEquals(num_items, num_magic_values);
  }
}

void TestForEach(thrust::default_random_engine &rng)
{
  TestForEach<int>(rng);
  TestForEach<std::size_t>(rng);
  TestForEachIterator();
  TestForEachOverwrite<hipcub::CacheLoadModifier::LOAD_DEFAULT>();
  TestForEachOverwrite<hipcub::CacheLoadModifier::LOAD_CA>();
}

int main(int argc, char **argv)
{
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  thrust::default_random_engine rng;

  TestBulk(rng);
  TestForEach(rng);
}
