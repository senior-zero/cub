#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockAdjacentDifference utilities
 ******************************************************************************/

#include <tuple>
#include <type_traits>

#define CATCH_CONFIG_RUNNER
#include <catch2/catch.hpp>

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <limits>
#include <memory>
#include <typeinfo>

#include <cub/block/block_adjacent_difference.cuh>
#include <cub/util_allocator.cuh>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/mismatch.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/sort.h>
#include <thrust/tabulate.h>

using namespace hipcub;

template <unsigned int ItemsPerThreadArg>
struct ThreadConfiguration
{
  constexpr static unsigned int ItemsPerThread = ItemsPerThreadArg;
};

template <unsigned int ThreadsInBlockArg>
struct ThreadBlockConfiguration
{
  constexpr static unsigned int ThreadsInBlock = ThreadsInBlockArg;
};

/**
 * \brief Generates integer sequence \f$S_n=i(i-1)/2\f$.
 *
 * The adjacent difference of this sequence produce consecutive numbers:
 * \f[
 *   p = \frac{i(i - 1)}{2} \\
 *   n = \frac{(i + 1) i}{2} \\
 *   n - p = i \\
 *   \frac{(i + 1) i}{2} - \frac{i (i - 1)}{2} = i \\
 *   (i + 1) i - i (i - 1) = 2 i \\
 *   (i + 1) - (i - 1) = 2 \\
 *   2 = 2
 * \f]
 */
template <typename DestT>
struct TestSequenceGenerator
{
  std::size_t offset;

  TestSequenceGenerator(std::size_t offset = 0)
      : offset(offset)
  {}

  template <typename SourceT>
  __device__ __host__ DestT operator()(SourceT index) const
  {
    index += static_cast<SourceT>(offset);
    return static_cast<DestT>(index * (index - 1) / SourceT(2));
  }
};

struct CustomType
{
  unsigned int key;
  unsigned int value;

  __device__ __host__ CustomType()
      : key(0)
      , value(0)
  {}

  __device__ __host__ CustomType(unsigned int key, unsigned int value)
      : key(key)
      , value(value)
  {}
};

__device__ __host__ bool operator==(const CustomType &lhs,
                                    const CustomType &rhs)
{
  return lhs.key == rhs.key && lhs.value == rhs.value;
}

__device__ __host__ bool operator!=(const CustomType &lhs,
                                    const CustomType &rhs)
{
  return !(lhs == rhs);
}

__device__ __host__ CustomType operator-(const CustomType &lhs,
                                         const CustomType &rhs)
{
  return CustomType{lhs.key - rhs.key, lhs.value - rhs.value};
}

struct CustomDifference
{
  template <typename DataType>
  __device__ DataType operator()(DataType &lhs, DataType &rhs)
  {
    return lhs - rhs;
  }
};

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void LastTileTestKernel(const DataType *input,
                                   DataType *output,
                                   unsigned int valid_items)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeftPartialTile(thread_result,
                               thread_data,
                               CustomDifference(),
                               valid_items);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRightPartialTile(thread_result,
                                thread_data,
                                CustomDifference(),
                                valid_items);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_result[item];
  }
}

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void MiddleTileTestKernel(const DataType *input,
                                     DataType *output,
                                     DataType neighbour_tile_value)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_result,
                    thread_data,
                    CustomDifference(),
                    neighbour_tile_value);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_result,
                     thread_data,
                     CustomDifference(),
                     neighbour_tile_value);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_result[item];
  }
}

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void MiddleTileInplaceTestKernel(const DataType *input,
                                            DataType *output,
                                            DataType neighbour_tile_value)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data,
                    thread_data,
                    CustomDifference(),
                    neighbour_tile_value);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data,
                     thread_data,
                     CustomDifference(),
                     neighbour_tile_value);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_data[item];
  }
}

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void TestKernel(DataType *data)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = data[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_result, thread_data, CustomDifference());
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_result, thread_data, CustomDifference());
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    data[thread_offset + item] = thread_result[item];
  }
}

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void LastTileTestInplaceKernel(const DataType *input,
                                          DataType *output,
                                          unsigned int valid_items)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeftPartialTile(thread_data,
                               thread_data,
                               CustomDifference(),
                               valid_items);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRightPartialTile(thread_data,
                                thread_data,
                                CustomDifference(),
                                valid_items);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_data[item];
  }
}

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void TestInplaceKernel(DataType *data)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = data[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data, thread_data, CustomDifference());
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data, thread_data, CustomDifference());
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    data[thread_offset + item] = thread_data[item];
  }
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void LastTileTest(const DataType *input,
                  DataType *output,
                  unsigned int valid_items)
{
  LastTileTestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, valid_items);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void Test(DataType *data)
{
  TestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(data);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void MiddleTileTest(const DataType *input,
                    DataType *output,
                    DataType neighbour_tile_value)
{
  MiddleTileTestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, neighbour_tile_value);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void LastTileInplaceTest(const DataType *input,
                         DataType *output,
                         unsigned int valid_items)
{
  LastTileTestInplaceKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, valid_items);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void InplaceTest(DataType *data)
{
  TestInplaceKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(data);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void MiddleTileInplaceTest(const DataType *input,
                           DataType *output,
                           DataType neighbour_tile_value)
{
  MiddleTileInplaceTestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, neighbour_tile_value);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}

template <typename FirstIteratorT, typename SecondOperatorT>
bool CheckResult(FirstIteratorT first_begin,
                 FirstIteratorT first_end,
                 SecondOperatorT second_begin)
{
  auto err = thrust::mismatch(first_begin, first_end, second_begin);

  if (err.first != first_end)
  {
    return false;
  }

  return true;
}

template <typename T>
struct Configuration256TB1IPT
{
  using Type = T;
  constexpr static unsigned int ThreadsInBlock = 256;
  constexpr static unsigned int ItemsPerThread = 1;
};

template <typename T>
struct Configuration256TB2IPT
{
  using Type = T;
  constexpr static unsigned int ThreadsInBlock = 256;
  constexpr static unsigned int ItemsPerThread = 2;
};

template <typename T>
struct Configuration256TB4IPT
{
  using Type = T;
  constexpr static unsigned int ThreadsInBlock = 256;
  constexpr static unsigned int ItemsPerThread = 4;
};

template <typename T>
struct Configuration128TB1IPT
{
  using Type = T;
  constexpr static unsigned int ThreadsInBlock = 128;
  constexpr static unsigned int ItemsPerThread = 1;
};

template <typename T>
struct Configuration128TB2IPT
{
  using Type = T;
  constexpr static unsigned int ThreadsInBlock = 128;
  constexpr static unsigned int ItemsPerThread = 2;
};

template <typename T>
struct Configuration128TB4IPT
{
  using Type = T;
  constexpr static unsigned int ThreadsInBlock = 128;
  constexpr static unsigned int ItemsPerThread = 4;
};


TEMPLATE_PRODUCT_TEST_CASE("BlockAdjacentDifference in last tile",
                           "[left][right]",
                           (Configuration128TB1IPT,
                            Configuration128TB2IPT,
                            Configuration128TB4IPT,
                            Configuration256TB1IPT,
                            Configuration256TB2IPT,
                            Configuration256TB4IPT),
                           (std::uint16_t, std::uint32_t, std::uint64_t))
{
  using DataType = typename TestType::Type;
  constexpr unsigned int ItemsPerThread = TestType::ItemsPerThread; // GENERATE doesn't return compile-time result
  constexpr unsigned int ThreadsInBlock = TestType::ThreadsInBlock;

  constexpr unsigned int tile_size = ItemsPerThread * ThreadsInBlock;
  thrust::device_vector<DataType> d_input(tile_size);

  for (bool inplace : {false, true})
  {
    for (unsigned int num_items = tile_size; num_items > 1; num_items /= 2)
    {
      thrust::tabulate(d_input.begin(),
                       d_input.end(),
                       TestSequenceGenerator<DataType>{});
      thrust::device_vector<DataType> d_output(d_input.size());

      constexpr bool read_left  = true;
      constexpr bool read_right = false;

      DataType *d_input_ptr  = thrust::raw_pointer_cast(d_input.data());
      DataType *d_output_ptr = thrust::raw_pointer_cast(d_output.data());

      SECTION( "calculating left adjacent difference" )
      {
        if (inplace)
        {
          LastTileInplaceTest<DataType,
                              ItemsPerThread,
                              ThreadsInBlock,
                              read_left>(d_input_ptr, d_output_ptr, num_items);
        }
        else
        {
          LastTileTest<DataType, ItemsPerThread, ThreadsInBlock, read_left>(
            d_input_ptr,
            d_output_ptr,
            num_items);
        }

        {
          using CountingIteratorT =
          typename thrust::counting_iterator<DataType,
            thrust::use_default,
            std::size_t,
            std::size_t>;

          REQUIRE( d_output.front() == d_input.front() );
          REQUIRE( CheckResult(d_output.begin() + 1,
                               d_output.begin() + num_items,
                               CountingIteratorT(DataType{0})));
          REQUIRE( CheckResult(d_output.begin() + num_items,
                               d_output.end(),
                               d_input.begin() + num_items));
        }
      }

      thrust::tabulate(d_input.begin(),
                       d_input.end(),
                       TestSequenceGenerator<DataType>{});

      SECTION( "calculating right adjacent difference" )
      {
        if (inplace)
        {
          LastTileInplaceTest<DataType,
                              ItemsPerThread,
                              ThreadsInBlock,
                              read_right>(d_input_ptr, d_output_ptr, num_items);
        }
        else
        {
          LastTileTest<DataType, ItemsPerThread, ThreadsInBlock, read_right>(
            d_input_ptr,
            d_output_ptr,
            num_items);
        }

        {
          thrust::device_vector<DataType> reference(num_items);
          thrust::sequence(reference.begin(),
                           reference.end(),
                           static_cast<DataType>(0),
                           static_cast<DataType>(-1));

          REQUIRE(CheckResult(d_output.begin(),
                              d_output.begin() + num_items - 1,
                              reference.begin()));
          REQUIRE(CheckResult(d_output.begin() + num_items - 1,
                              d_output.end(),
                              d_input.begin() + num_items - 1));
        }
      }
    }
  }
}

int main(int argc, char *argv[])
{
  // global setup...

  int result = Catch::Session().run(argc, argv);

  // global clean-up...

  return result;
}